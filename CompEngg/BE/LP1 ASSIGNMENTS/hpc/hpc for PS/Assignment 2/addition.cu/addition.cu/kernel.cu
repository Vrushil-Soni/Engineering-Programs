#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 512


__global__ void add(int* a, int* b, int* c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void)
{
	int* a, * b, * c;
	int* dev_a, * dev_b, * dev_c;
	int size = N * sizeof(int);
	int i;


	hipMalloc((void**)& dev_a, size);
	hipMalloc((void**)& dev_b, size);
	hipMalloc((void**)& dev_c, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	for (i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i;
	}

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	add << <N, 1 >> > (dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);


	for (i = 0; i < 10; i++)
	{
		printf("addition of %ds is %d\n", i, c[i]);
	}

	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
/*
addition of 0s is 0
addition of 1s is 2
addition of 2s is 4
addition of 3s is 6
addition of 4s is 8
addition of 5s is 10
addition of 6s is 12
addition of 7s is 14
addition of 8s is 16
addition of 9s is 18

C:\Users\Vrushil\Desktop\ProgramSupplier\HPC running\addition.cu\x64\Debug\addition.cu.exe (process 4140) exited with code 0.
Press any key to close this window . . .

*/
