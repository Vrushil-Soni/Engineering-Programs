#include "hip/hip_runtime.h"
/* Matrix Multiplication in CUDA */
#include<stdio.h>
__global__ void matrix_mul(float* ad, float* bd, float* cd, int N) {
	float pvalue = 0;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	int Col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = 0; i < N; ++i) {
		float m = ad[Row * N + i];
		float n = bd[i * N + Col];
		pvalue += m * n;
	}

	cd[Row * N + Col] = pvalue;

}
int main() {
	int N = 16, i, j;				//N == size of square matrix
	float* a, * b, * c;
	float* ad, * bd, * cd;

	FILE* f;
	f = fopen("Parallel Multiply.txt", "w");

	size_t size = sizeof(float) * N * N;

	a = (float*)malloc(size);
	b = (float*)malloc(size);
	c = (float*)malloc(size);

	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			a[i * N + j] = 1.0; //(float)(i*N+j);	//initializing each value with its own index
			b[i * N + j] = 1.0; //(float)(i*N+j);		//random functions can be used alternatively
		}
	}

	hipMalloc(&ad, size);
	hipMalloc(&bd, size);
	hipMalloc(&cd, size);

	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

	dim3 blocksize(8, 8);		    //each block contains 16 * 16 (=256) threads
	dim3 gridsize(N / 8, N / 8);		//creating just sufficient no of blocks

	matrix_mul << <gridsize, blocksize >> > (ad, bd, cd, N);


	hipMemcpy(c, cd, sizeof(float) * N * N, hipMemcpyDeviceToHost);

	fprintf(f, "Array A was---\n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++)
			fprintf(f, "%f ", a[i * N + j]);
		fprintf(f, "\n");
	}
	fprintf(f, "\nArray B was---\n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++)
			fprintf(f, "%f ", b[i * N + j]);
		fprintf(f, "\n");
	}
	fprintf(f, "\nMultiplication of A and B gives C----\n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++)
			fprintf(f, "%f ", c[i * N + j]); //if correctly computed, then all values must be N
		fprintf(f, "\n");
	}
	printf("\nYou can see output in Parallel Mutiply.txt file in project directory");

	fclose(f);

	hipDeviceReset();
	free(a);
	free(b);
	free(c);
	return 1;
}

/*

You can see output in Parallel Mutiply.txt file in project directory
C:\Users\Vrushil\Desktop\ProgramSupplier\HPC running\Assignment2\matrix_mult_file\x64\Debug\matrix_mult_file.exe (process 17716) exited with code 1.
Press any key to close this window . . .

*/