#include "hip/hip_runtime.h"
/* Matrix Multiplication in CUDA */
#include<stdio.h>
__global__ void matrix_mul(float* ad, float* bd, float* cd, int N) {
	float pvalue = 0;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	int Col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = 0; i < N; ++i) {
		float m = ad[Row * N + i];
		float n = bd[i * N + Col];
		pvalue += m * n;
	}

	cd[Row * N + Col] = pvalue;

}
int main() {
	int N = 8, i, j;				//N == size of square matrix
	float* a, * b, * c;
	float* ad, * bd, * cd;

	FILE* f;
	f = fopen("Parallel Multiply.txt", "w");

	size_t size = sizeof(float) * N * N;

	a = (float*)malloc(size);
	b = (float*)malloc(size);
	c = (float*)malloc(size);

	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			a[i * N + j] = 1.0; //(float)(i*N+j);	//initializing each value with its own index
			b[i * N + j] = 1.0; //(float)(i*N+j);	//random functions can be used alternatively
		}
	}
	hipMalloc(&ad, size);
	hipMalloc(&bd, size);
	hipMalloc(&cd, size);

	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
	printf("\nAfter HostToDevice Memcpy\n%s\n",
		hipGetErrorString(hipGetLastError()));

	dim3 blocksize(8, 8);		    //each block contains 16 * 16 (=256) threads
	dim3 gridsize(N / 8, N / 8);		//creating just sufficient no of blocks

	matrix_mul << <gridsize, blocksize >> > (ad, bd, cd, N);

	hipMemcpy(c, cd, sizeof(float) * N * N, hipMemcpyDeviceToHost);
	printf("\nAfter DeviceToHost Memcpy\n%s\n",
		hipGetErrorString(hipGetLastError()));

	printf("Array A was---\n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++)
			printf("%f ", a[i * N + j]);
		printf("\n");
	}
	printf("\nArray B was---\n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++)
			printf("%f ", b[i * N + j]);
		printf("\n");
	}
	printf("\nMultiplication of A and B gives C----\n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++)
			printf("%f ", c[i * N + j]); //if correctly computed, then all values must be N
		printf("\n");
	}
	printf("\nYou can see output in Parallel Mutiply.txt file in project directory");

	fclose(f);

	free(a);
	free(b);
	free(c);
	return 1;
}
/*
After HostToDevice Memcpy
no error

After DeviceToHost Memcpy
no error
Array A was---
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000

Array B was---
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000
1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000 1.000000

Multiplication of A and B gives C----
8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000
8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000
8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000
8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000
8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000
8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000
8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000
8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000 8.000000

You can see output in Parallel Mutiply.txt file in project directory
C:\Users\Vrushil\Desktop\ProgramSupplier\HPC running\Assignment2\matrix_mult_console\x64\Debug\matrix_mult_console.exe (process 15256) exited with code 1.
Press any key to close this window . . .
*/


