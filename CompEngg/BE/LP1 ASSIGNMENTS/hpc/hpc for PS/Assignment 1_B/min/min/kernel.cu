#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define SIZE 9


__global__ void min(int* a, int* c)
{
	int i = threadIdx.x;

	*c = a[0];

	if (a[i] < *c)
	{
		*c = a[i];
	}

}

int main()
{
	int i;


	int a[SIZE];
	int c;

	int* dev_a, * dev_c;

	hipMalloc((void**)& dev_a, SIZE * sizeof(int));
	hipMalloc((void**)& dev_c, SIZE * sizeof(int));

	cout << "Enter the numbers  : \n";
	for (i = 0; i < SIZE; i++)
	{
		cin >> a[i];
	}
	for (i = 0; i < SIZE; i++)
	{
		cout << a[i] << " ";
	}

	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	min <<<1, SIZE >>> (dev_a, dev_c);
	hipMemcpy(&c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	cout << "\n min value = ";
	cout << c;

	hipFree(dev_a);
	hipFree(dev_c);


	return 0;
}
/*
Enter the numbers  :
10
54
23
7
4
32
33
15
40
10 54 23 7 4 32 33 15 40
 min value = 4
C:\Users\Vrushil\Desktop\ProgramSupplier\HPC running\Assignment1B\min\x64\Debug\min.exe (process 17564) exited with code 3.
Press any key to close this window . . .

*/
