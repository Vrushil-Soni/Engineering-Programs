#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define SIZE 7


__global__ void max(int* a, int* c)
{
	int i = threadIdx.x;

	*c = a[0];

	if (a[i] > * c)
	{
		*c = a[i];
	}

}

int main()
{
	int i;


	int a[SIZE];
	int c;

	int* dev_a, * dev_c;

	hipMalloc((void**)& dev_a, SIZE * sizeof(int));
	hipMalloc((void**)& dev_c, SIZE * sizeof(int));

	cout << "Enter the numbers  : \n";
	for (i = 0; i < SIZE; i++)
	{
		cin >> a[i];
	}
	for (i = 0; i < SIZE; i++)
	{
		cout << a[i] << " ";
	}

	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	max << <1, SIZE >> > (dev_a, dev_c);
	hipMemcpy(&c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	cout << "\n max value = ";
	cout << c;

	hipFree(dev_a);
	hipFree(dev_c);


	return 0;
}
/*
Enter the numbers  :
1
5
3
2
7
9
4
1 5 3 2 7 9 4
 max value = 4
C:\Users\Vrushil\Desktop\ProgramSupplier\HPC running\Assignment1B\max_min\x64\Debug\max_min.exe (process 14076) exited with code 3.
Press any key to close this window . . .
*/